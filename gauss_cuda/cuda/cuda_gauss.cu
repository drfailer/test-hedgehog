#include "hip/hip_runtime.h"
#include "cuda_gauss.h"
#include <cstdio>

__global__ void substractLines_impl(double *res, double *pivot, double *resVector,
        double *pivotVector, double coef, size_t size) {
    size_t i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < size) {
        res[i] -= pivot[i] * coef;
    }
    if (i == 0) {
        *resVector -= *pivotVector * coef;
    }
}

void substractLines(double *res, double *pivot, double *resVector,
        double *pivotVector, double coef, size_t size, hipStream_t stream)
{
    int threadsPerBlock = 256;
    int blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;
    substractLines_impl<<<blocksPerGrid, threadsPerBlock, 0, stream>>>(
            res, pivot, resVector, pivotVector, coef, size);
}
